#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "basic_blur.h"

static uint8_t * dev_src, * dev_dst;

__global__ void kernGaussianBlur(int width, int height, uint8_t * dst, uint8_t * src) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x >= width || y >= height) {
		return;
	}
	float kernel[5][5] = {
    {0.003765, 0.015019, 0.023792, 0.015019, 0.003765},
    {0.015019, 0.059912, 0.094907, 0.059912, 0.015019},
    {0.023792, 0.094907, 0.150342, 0.094907, 0.023792},
    {0.015019, 0.059912, 0.094907, 0.059912, 0.015019},
    {0.003765, 0.015019, 0.023792, 0.015019, 0.003765}
  };
	float r, g, b;
	r = g = b = 0.0;
	for (int i = 0; i < 5; i++) {
		int tx = x + i - 2;
		for (int j = 0; j < 5; j++) {
			int ty = y + j - 2;
			if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
				r += src[(ty * width + tx) * 3] * kernel[i][j];
				g += src[(ty * width + tx) * 3 + 1] * kernel[i][j];
				b += src[(ty * width + tx) * 3 + 2] * kernel[i][j];
			}
		}
	}
	int idx = 3 * (y * width + x);
	dst[idx] = r;
	dst[idx + 1] = g;
	dst[idx + 2] = b;
	return;
}
void init(int width, int height) {
	int sz = sizeof(uint8_t) * width * height * 3;
	hipMalloc(&dev_src, sz);
	hipMalloc(&dev_dst, sz);
}

void cleanup() {
	hipFree(dev_src);
	dev_src = NULL;
	hipFree(dev_dst);
	dev_dst = NULL;
}

void blurFrame(uint8_t * dst, uint8_t * src, int width, int height) {
	int sz = sizeof(uint8_t) * width * height * 3;
	hipMemcpy(dev_src, src, sz, hipMemcpyHostToDevice);
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(width + blockSize2d.x - 1) / blockSize2d.x,
		(height + blockSize2d.y - 1) / blockSize2d.y);
	
	kernGaussianBlur<<<blocksPerGrid2d, blockSize2d>>>(width, height, dev_dst, dev_src);
	hipDeviceSynchronize();
	hipMemcpy(dst, dev_dst, sz, hipMemcpyDeviceToHost);
	return;
}
