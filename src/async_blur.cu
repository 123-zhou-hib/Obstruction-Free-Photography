#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "basic_blur.h"

static uint8_t * dev_src[3];
static uint8_t * dev_dst[3];
static int cur = 0;
static hipStream_t uploadStream, downloadStream, computeStream;

__global__ void kernGaussianBlur(int width, int height, uint8_t * dst, uint8_t * src) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (x >= width || y >= height) {
		return;
	}
	float kernel[5][5] = {
    {0.003765, 0.015019, 0.023792, 0.015019, 0.003765},
    {0.015019, 0.059912, 0.094907, 0.059912, 0.015019},
    {0.023792, 0.094907, 0.150342, 0.094907, 0.023792},
    {0.015019, 0.059912, 0.094907, 0.059912, 0.015019},
    {0.003765, 0.015019, 0.023792, 0.015019, 0.003765}
  };
	float r, g, b;
	r = g = b = 0.0;
	for (int i = 0; i < 5; i++) {
		int tx = x + i - 2;
		for (int j = 0; j < 5; j++) {
			int ty = y + j - 2;
			if (tx >= 0 && ty >= 0 && tx < width && ty < height) {
				r += src[(ty * width + tx) * 3] * kernel[i][j];
				g += src[(ty * width + tx) * 3 + 1] * kernel[i][j];
				b += src[(ty * width + tx) * 3 + 2] * kernel[i][j];
			}
		}
	}
	int idx = 3 * (y * width + x);
	dst[idx] = r;
	dst[idx + 1] = g;
	dst[idx + 2] = b;
	return;
}
void init(int width, int height) {
	int sz = sizeof(uint8_t) * width * height * 3;
	for (int i = 0; i < 3; i++) {
		hipMalloc(&dev_src[i], sz);
		hipMalloc(&dev_dst[i], sz);
	}
	hipStreamCreate(&uploadStream);
	hipStreamCreate(&computeStream);
	hipStreamCreate(&downloadStream);
}

void cleanup() {
	for (int i = 0; i < 3; i++) {
		hipFree(dev_src[i]);
		dev_src[i] = NULL;
		hipFree(dev_dst[i]);
		dev_dst[i] = NULL;
	}
}

void blurFrame(uint8_t * dst, uint8_t * src, int width, int height) {
	int sz = sizeof(uint8_t) * width * height * 3;
	hipMemcpyAsync(dev_src[cur % 3], src, sz, hipMemcpyHostToDevice, uploadStream);
	const dim3 blockSize2d(8, 8);
	const dim3 blocksPerGrid2d(
		(width + blockSize2d.x - 1) / blockSize2d.x,
		(height + blockSize2d.y - 1) / blockSize2d.y);
	
	if (cur >= 1) {
		kernGaussianBlur<<<blocksPerGrid2d, blockSize2d, 0, computeStream>>>(
			width, height, dev_dst[(cur - 1) % 3], dev_src[(cur + 1) % 3]);
	}
	if (cur >= 2) {
		hipMemcpyAsync(dst, dev_dst[(cur - 2) % 3], sz, hipMemcpyDeviceToHost, downloadStream);
	}
	hipDeviceSynchronize();
	cur++;
	return;
}
